#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"
#include <hip/hip_runtime_api.h>



//compile with c++ lodepng file
//nvcc boxBlur.cu lodepng.cpp

__global__ void boxBlur(unsigned char *ImageInput, unsigned char * ImageOuput, int width, int height){

	int filter[] = {NULL,NULL,NULL,NULL,NULL,NULL,NULL,NULL,NULL};

	int red=0;
	int green=0;
	int blue=0;
	int transperency=0;

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i==0){
		filter[0] = i;
		filter[1] = i+1;
		filter[2] = i+width;
		filter[3] = i+width+1;

	}
	else if(i>0 && i<(width-1)){
		filter[0] = i;
		filter[1] = i+1;
		filter[2] = i-1;
		filter[3] = i+width;
		filter[4] = 1+i+width;
		filter[5] = i+width-1;
	}
	else if (i==(width-1)){
		filter[0] = i;
		filter[1] = i-1;
		filter[2] = i+width;
		filter[3] = i+width-1;
	}
	else if(((i > width-1 && i< (height*width)-width) && ((i+1) % width ==0))){
		filter[0] = i;
		filter[1] = i-1;
		filter[2] = i-width;
		filter[3] = i-width-1;
		filter[4] = i+width;
		filter[5] = i+width-1;
	}
	else if (i==((height*width)-1)){
		filter[0] = i;
		filter[1] = i-1;
		filter[2] = i-width-1;
		filter[3] = i-width;
	}
	else if(i>((height*width)-width) && i < (height*width)){
		filter[0] = i;
		filter[1] = i+1;
		filter[2] = i-1;
		filter[3] = i-width;
		filter[4] = i-width-1;
		filter[5] = i-width+1;
	}
	else if(i==(height*width)-width){
		filter[0] = i;
		filter[1] = i+1;
		filter[2] = i-width;
		filter[3] = i-width+1;
	}
	else if((i>width-1 &&i<(height*width)-(2*width+1))&&i % width ==0){
		filter[0] = i;
		filter[1] = i+1;
		filter[2] = i+width;
		filter[3] = i+width+1;
		filter[4] = i-width;
		filter[5] = i-width+1;

	}
	else{
		filter[0] = i;
		filter[1] = i+1;
		filter[2] = i-1;
		filter[3] = i+width;
		filter[4] = i+width+1;
		filter[5] = i+width-1;
		filter[6] = i-width;
		filter[7] = i-width+1;
		filter[8] = i-width-1;
	}



	int pixel = i*4;
	int c=0;
	for (int i=0; i<sizeof(filter)/sizeof(filter[0]); i++){
		if(filter[i] != NULL){
			red += ImageInput[filter[i]*4];
			green += ImageInput[filter[i]*4+1];
			blue += ImageInput[filter[i]*4+2];
			c++;
		}
	}
	
	red = red/c;
	green = green/c;
	blue = blue/c;
	transperency = ImageInput[i*4+3];


	ImageOuput[pixel] = red;
	ImageOuput[1+pixel] = green;
	ImageOuput[2+pixel] = blue;
	ImageOuput[3+pixel] = transperency;
}



int main () {
    // png decode variables
    unsigned char *img;
    unsigned int height, width;

    // lodepng decode
    lodepng_decode32_file(&img, &width, &height, "hck.png");

	printf("Width: %d\nHeight: %d\n", width, height);

    // image pixel's values RGBT and its space on memory
    int totalValues = height * width * 4;
    int totalSpace = totalValues * sizeof(unsigned char);

    // dynamic memory allocation for image data in host
    unsigned char *hostImageInput;
    hostImageInput = (unsigned char *) malloc(totalSpace);
    unsigned char *hostImageOutput;
    hostImageOutput = (unsigned char *) malloc(totalSpace);


    // PUT IMAGE DATA INSIDE host input image array
    for (int i=0; i<totalValues; i++) {
        hostImageInput[i]  = img[i];
    }

    // declare memory pointers for GPU (device)
    unsigned char *deviceImageInput;
    unsigned char *deviceImageOutput;


    // allocate memory for gpu
    hipMalloc( (void **) &deviceImageInput, totalSpace);
    hipMalloc( (void **) &deviceImageOutput, totalSpace);


    // allocate memory in gpu with image data
    hipMemcpy(deviceImageInput, hostImageInput, totalSpace, hipMemcpyHostToDevice);


    // invoke kernel function that runs on device and blur the image with 3x3 matrix approach
    boxBlur<<<height, width>>>(deviceImageInput, deviceImageOutput, width, height);

    // copy device image data (blurred) to host image array
    hipMemcpy(hostImageOutput, deviceImageOutput, totalSpace, hipMemcpyDeviceToHost);
	
    // encode output image data to new image
	lodepng_encode32_file("output.png", hostImageOutput, width, height);


    // free dynamic allocation
    free(img);
    free(hostImageInput);
    free(hostImageOutput);
    hipFree(deviceImageInput);
    hipFree(deviceImageOutput);

    return 0;
}