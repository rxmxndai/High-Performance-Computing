#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"
#include <hip/hip_runtime_api.h>



//compile with c++ lodepng file
//nvcc boxBlur.cu lodepng.cpp

__global__ void boxBlur(unsigned char *ImageInput, unsigned char * ImageOuput, int width, int height){

	int filter[] = {NULL,NULL,NULL,NULL,NULL,NULL,NULL,NULL,NULL};

	int red=0;
	int green=0;
	int blue=0;
	int transperency=0;

	int index = blockDim.x * blockIdx.x + threadIdx.x;


	if(index==0){
		filter[0] = index;
		filter[1] = index+1;
		filter[2] = index+width;
		filter[3] = index+width+1;

	}
	else if(index>0 && index<(width-1)){
		filter[0] = index;
		filter[1] = index+1;
		filter[2] = index-1;
		filter[3] = index+width;
		filter[4] = 1+index+width;
		filter[5] = index+width-1;
	}
	else if (index==(width-1)){
		filter[0] = index;
		filter[1] = index-1;
		filter[2] = index+width;
		filter[3] = index+width-1;
	}
	else if(((index > width-1 && index< (height*width)-width) && ((index+1) % width ==0))){
		filter[0] = index;
		filter[1] = index-1;
		filter[2] = index-width;
		filter[3] = index-width-1;
		filter[4] = index+width;
		filter[5] = index+width-1;
	}
	else if (index==((height*width)-1)){
		filter[0] = index;
		filter[1] = index-1;
		filter[2] = index-width-1;
		filter[3] = index-width;
	}
	else if(index>((height*width)-width) && index < (height*width)){
		filter[0] = index;
		filter[1] = index+1;
		filter[2] = index-1;
		filter[3] = index-width;
		filter[4] = index-width-1;
		filter[5] = index-width+1;
	}
	else if(index==(height*width)-width){
		filter[0] = index;
		filter[1] = index+1;
		filter[2] = index-width;
		filter[3] = index-width+1;
	}
	else if((index>width-1 &&index<(height*width)-(2*width+1))&&index % width ==0){
		filter[0] = index;
		filter[1] = index+1;
		filter[2] = index+width;
		filter[3] = index+width+1;
		filter[4] = index-width;
		filter[5] = index-width+1;

	}
	else{
		filter[0] = index;
		filter[1] = index+1;
		filter[2] = index-1;
		filter[3] = index+width;
		filter[4] = index+width+1;
		filter[5] = index+width-1;
		filter[6] = index-width;
		filter[7] = index-width+1;
		filter[8] = index-width-1;
	}



	int pixel = index*4;
	int c=0;
	for (int index=0; index<sizeof(filter)/sizeof(filter[0]); index++){
		if(filter[index] != NULL){
			red += ImageInput[filter[index]*4];
			green += ImageInput[filter[index]*4+1];
			blue += ImageInput[filter[index]*4+2];
			c++;
		}
	}
	

	red = red/c;
	green = green/c;
	blue = blue/c;
	transperency = ImageInput[index*4+3];


	ImageOuput[pixel] = red;
	ImageOuput[1+pixel] = green;
	ImageOuput[2+pixel] = blue;
	ImageOuput[3+pixel] = transperency;
	
}



int main () {
    // png decode variables
    unsigned char *img;
    unsigned int height, width;

    // lodepng decode
    lodepng_decode32_file(&img, &width, &height, "hck.png");

	printf("Width: %d\nHeight: %d\n", width, height);

    // image pixel's values RGBT and its space on memory
    int totalValues = height * width * 4;
    int totalSpace = totalValues * sizeof(unsigned char);

    // dynamic memory allocation for image data in host
    unsigned char *hostImageInput;
    hostImageInput = (unsigned char *) malloc(totalSpace);
    unsigned char *hostImageOutput;
    hostImageOutput = (unsigned char *) malloc(totalSpace);


    // PUT IMAGE DATA INSIDE host input image array
    for (int index=0; index<totalValues; index++) {
        hostImageInput[index]  = img[index];
    }

    // declare memory pointers for GPU (device)
    unsigned char *deviceImageInput;
    unsigned char *deviceImageOutput;


    // allocate memory for gpu
    hipMalloc( (void **) &deviceImageInput, totalSpace);
    hipMalloc( (void **) &deviceImageOutput, totalSpace);


    // allocate memory in gpu with image data
    hipMemcpy(deviceImageInput, hostImageInput, totalSpace, hipMemcpyHostToDevice);


    // invoke kernel function that runs on device and blur the image with 3x3 matrix approach
    boxBlur<<<height, width>>>(deviceImageInput, deviceImageOutput, width, height);

    // copy device image data (blurred) to host image array
    hipMemcpy(hostImageOutput, deviceImageOutput, totalSpace, hipMemcpyDeviceToHost);
	
    // encode output image data to new image
	lodepng_encode32_file("cudaBlur.png", hostImageOutput, width, height);


    // free dynamic allocation
    free(img);
    free(hostImageInput);
    free(hostImageOutput);
    hipFree(deviceImageInput);
    hipFree(deviceImageOutput);

    return 0;
}